#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void myKernel(int n) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < n) printf("Hello from thread %d\n", idx);
}
